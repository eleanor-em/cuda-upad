#include "hip/hip_runtime.h"
#ifndef _KERNEL_CU
#define _KERNEL_CU

#include <chrono>
#include "memory.cu"
#include "unionfind.cu"

void finalise_kernel() {
    auto result = hipDeviceSynchronize();
    if (result != 0) {
        printf(hipGetErrorName(result));
    }
}

class Timer {
    public:
        Timer() {
            reset();
        }

        void reset() {
            start = std::chrono::high_resolution_clock::now();
        }

        void report(const char *name) {
            auto end = std::chrono::high_resolution_clock::now();
            std::chrono::duration<double> diff = end - start;
            printf("%s ran in %fms\n", name, diff.count() * 1000);
        }
    private:
        std::chrono::time_point<std::chrono::high_resolution_clock> start;
};

template<size_t block_size>
__global__ void local_ccl(DevicePointer<uint8_t>::Raw image, uint16_t image_width, DevicePointer<uint32_t>::Raw labelmap) {
    uint16_t tid = threadIdx.x + threadIdx.y * blockDim.x;
    uint16_t x = blockDim.x * blockIdx.x + threadIdx.x;
    uint16_t y = blockDim.y * blockIdx.y + threadIdx.y;

    // Initialise shared data
    __shared__ uint16_t labelset[block_size * block_size];
    __shared__ uint8_t subimage[block_size * block_size];
    make_set(labelset, tid);
    subimage[tid] = image[y * image_width + x];
    __syncthreads();

    // Scan rows
    if (tid > 0 && subimage[tid] == subimage[tid - 1]) {
        merge(labelset, tid, (uint16_t) (tid - 1));
    }
    __syncthreads();

    // Scan columns
    if (tid >= block_size && subimage[tid] == subimage[tid - block_size]) {
        merge(labelset, tid, (uint16_t) (tid - block_size));
    }
    __syncthreads();

    // Perform labelling
    uint32_t l = find_root(labelset, tid);
    l = blockIdx.x * blockDim.x + l % blockDim.x + (blockIdx.y * blockDim.y + l / blockDim.x) * image_width;
    labelmap[y * image_width + x] = l;
}

__global__ void seam_stitch(DevicePointer<uint8_t>::Raw image, uint16_t image_width, DevicePointer<uint32_t>::Raw labelmap) {
    // Stitch row seams
    uint16_t x = (blockIdx.x + 1) * blockDim.x;
    uint16_t y = blockIdx.y * blockDim.x + threadIdx.x;
    uint32_t dest = y * image_width + x;
    uint32_t src = dest - 1;
    if (image[dest] == image[src]) {
        merge(labelmap, dest, src);
    }
    __syncthreads();

    // Stitch column seams
    x = blockIdx.y * blockDim.x + threadIdx.x;
    y = (blockIdx.x + 1) * blockDim.x;
    dest = y * image_width + x;
    src = (y - 1) * image_width + x;
    if (image[dest] == image[src]) {
        merge(labelmap, dest, src);
    }
}

__global__ void find_all_roots(DevicePointer<uint32_t>::Raw labelmap) {
    uint32_t tid = (blockIdx.x + gridDim.x * blockIdx.y) * blockDim.x * blockDim.y + threadIdx.x + blockDim.x * threadIdx.y;
    labelmap[tid] = find_root(labelmap, tid);
}

HostPointer<uint32_t> label_components(const DevicePointer<uint8_t>& image, uint16_t width, uint16_t height) {
    const uint8_t block_size = 32;
    const uint16_t blocks_per_row = width > block_size ? width / block_size : 1;
    const dim3 image_grid(blocks_per_row, blocks_per_row);
    const dim3 image_block(block_size, block_size);
    const dim3 seam_grid(blocks_per_row - 1, blocks_per_row);
    const dim3 seam_block(block_size, block_size);

    DevicePointer<uint32_t> labelmap(width * height);

    // Locally label connected components
    Timer timer, total_timer;
    local_ccl<block_size><<<image_grid, image_block>>>(image.as_raw(), width, labelmap.as_raw());
    finalise_kernel();
    timer.report("local-CCL kernel");

    // Stitch seams together
    timer.reset();
    seam_stitch<<<seam_grid, seam_block>>>(image.as_raw(), width, labelmap.as_raw());
    finalise_kernel();
    timer.report("seam-stitch kernel");

    // Find the root of all components
    timer.reset();
    find_all_roots<<<image_grid, image_block>>>(labelmap.as_raw());
    timer.report("find-all-roots kernel");

    total_timer.report("total");

    return labelmap.as_host();
}
#endif // _KERNEL_CU